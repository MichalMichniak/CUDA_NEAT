#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <cstdlib>
#include <cmath>
#include <hiprand/hiprand_kernel.h>

#define WIDTH 800
#define HEIGHT 600
#define GAP_R 75
#define FLOPPY_RADIUS 10

#define MAX_ITERATION_GOAL 100000

#define POPULATION_COUNT 1000
#define SURVIVORS_TOURNAMENTS 600
#define K 200
#define BLOCK_SIZE 128


#define MUTATION_T1 12
#define MUTATION_T2 35
#define MUTATION_W_P 0.05
int next_node_innov = 8;
int next_edge_innov = 26;

#define BLOCK_SIZE 128
// ###### CUMULATED HISTOGRAM BEGIN #######
#define SECTION_SIZE 512

__global__ void scanKernelX_with_copy(int *Y, int *Y_copy, int *S, int *X, int width)
{
    //@@ INSERT CODE HERE
    __shared__ int in[SECTION_SIZE];
    if(threadIdx.x + (blockDim.x * blockIdx.x) < width){
        in[threadIdx.x] = X[threadIdx.x + (blockDim.x * blockIdx.x)];
    }else{
        in[threadIdx.x] = 0;
    }
    __syncthreads();
    
    int result = 0;
    for(int start = 1; ((start + (blockDim.x * blockIdx.x)) < width) && (start < blockDim.x); start=start*2){
        if(threadIdx.x>=start){
            result = in[threadIdx.x] + in[threadIdx.x-start];
            // printf("it: %d\t(%d, %d)\t%f\t%f\t%f\n",it,threadIdx.x, threadIdx.x+(1<<it) ,in[threadIdx.x] ,in[threadIdx.x+(1<<it)],result);
        }
        __syncthreads();
        if(threadIdx.x>=start){
            in[threadIdx.x] = result;
        }
        //printf("%f\n",in[0]);
        __syncthreads();
    }
    
    if(threadIdx.x + (blockDim.x * blockIdx.x) < width){
        Y[threadIdx.x + (blockDim.x * blockIdx.x)] = in[threadIdx.x]; 
        Y_copy[threadIdx.x + (blockDim.x * blockIdx.x)] = in[threadIdx.x];
        if(threadIdx.x == (blockDim.x - 1)){
            S[blockIdx.x] = in[threadIdx.x];
        }
    }
    
}

__global__ void scanKernelS_with_copy(int *S, int width)
{
    //@@ INSERT CODE HERE
    __shared__ int in[SECTION_SIZE];
    if(threadIdx.x + (blockDim.x * blockIdx.x) < width){
        in[threadIdx.x] = S[threadIdx.x + (blockDim.x * blockIdx.x)];
    }else{
        in[threadIdx.x] = 0;
    }
    __syncthreads();

    int result = 0;
    for(int start = 1; start < width; start=start*2){
        if(threadIdx.x>=start){
            result = in[threadIdx.x] + in[threadIdx.x-start];
            // printf("it: %d\t(%d, %d)\t%f\t%f\t%f\n",it,threadIdx.x, threadIdx.x+(1<<it) ,in[threadIdx.x] ,in[threadIdx.x+(1<<it)],result);
        }
        __syncthreads();
        if(threadIdx.x>=start){
            in[threadIdx.x] = result;
        }
        //printf("%f\n",in[0]);
        __syncthreads();
    }
    
    if(threadIdx.x < width){
        S[threadIdx.x] = in[threadIdx.x]; 
    }
}

__global__ void updateYKernel_with_copy(int *Y, int *Y_copy, int *S, int widthY)
{
    //@@ INSERT CODE HERE
    if((blockIdx.x >= 1) && (threadIdx.x + (blockDim.x * blockIdx.x) < widthY)){
        Y[threadIdx.x + (blockDim.x * blockIdx.x)] += S[blockIdx.x - 1];
        Y_copy[threadIdx.x + (blockDim.x * blockIdx.x)] = Y[threadIdx.x + (blockDim.x * blockIdx.x)];
    }
}

void cumulatedHistogram_with_copy(int *d_Y, int *d_Y_copy, int *d_X, int width)
{
    /*
    input:
        device vectors:
            d_X - input
            d_Y - output
        width - size of vectors
    */
    int *d_S;
    hipMalloc(&d_S, ceil((float)width/SECTION_SIZE) * sizeof(int));

    dim3 dimGrid(ceil((float)width/SECTION_SIZE),1,1);
    dim3 dimBlock(SECTION_SIZE,1,1);
    
    scanKernelX_with_copy<<<dimGrid,dimBlock>>>(d_Y, d_Y_copy, d_S, d_X, width);

    dim3 dimGrid2(1,1,1);
    dim3 dimBlock2(SECTION_SIZE,1,1);

    scanKernelS_with_copy<<<dimGrid2,dimBlock2>>>(d_S, ceil((float)width/SECTION_SIZE));
    
    updateYKernel_with_copy<<<dimGrid,dimBlock>>>(d_Y, d_Y_copy, d_S, width);

    hipFree(d_S);
}
// ###### CUMULATED HISTOGRAM END #######

// ###### Reduction boolean count BEGIN #######

int reductionSequential_count_col_size(int *input, int width)
{
    int sum = 0;
    for (int i = 0; i < width; ++i)
    {
        sum += input[i];
    }

    return sum;
}

__global__ void reductionKernelOp_count_col_size(bool *input, int *output, int width)
{
    //@@ INSERT CODE HERE
    __shared__ int in[BLOCK_SIZE];
    if(threadIdx.x + (blockDim.x * blockIdx.x) < width){
        in[threadIdx.x] = (int)(input[threadIdx.x + (blockDim.x * blockIdx.x)]);
    }else{
        in[threadIdx.x] = 0;
    }
    __syncthreads();
    int targetlevel = 0;
    int index = BLOCK_SIZE;
    while (index >>= 1) ++targetlevel;

    int result = 0;
    for(int it = 0; it < targetlevel; it++){
        if(threadIdx.x<(BLOCK_SIZE/(2<<it))){
            result = in[threadIdx.x] + in[threadIdx.x+(BLOCK_SIZE/(2<<it))];
            // printf("it: %d\t(%d, %d)\t%f\t%f\t%f\n",it,threadIdx.x, threadIdx.x+(1<<it) ,in[threadIdx.x] ,in[threadIdx.x+(1<<it)],result);
        }
        __syncthreads();
        if((threadIdx.x<(BLOCK_SIZE/(2<<it)))){
            in[threadIdx.x] = result;
        }
        //printf("%f\n",in[0]);
        __syncthreads();
    }
    
    if(threadIdx.x == 0){
        output[blockIdx.x] = in[0]; 
        // printf("%f",in[0]);
        // printf("\n%d\n",targetlevel);
    }
    
}

int count_col_size(bool* d_enable, int length)
{
    //@@ INSERT CODE HERE
    int *d_output, *h_output;
    hipMalloc(&d_output, ceil((float)length/BLOCK_SIZE) * sizeof(int));
    h_output = (int*)malloc(ceil((float)length/BLOCK_SIZE) * sizeof(int));

    dim3 dimGrid(ceil((float)length/BLOCK_SIZE),1,1);
    dim3 dimBlock(BLOCK_SIZE,1,1);
    
    reductionKernelOp_count_col_size<<<dimGrid,dimBlock>>>(d_enable,d_output,length);
    
    hipMemcpy(h_output, d_output,  ceil((float)length/BLOCK_SIZE)*sizeof(int), hipMemcpyDeviceToHost);

    int result = reductionSequential_count_col_size(h_output,ceil((float)length/BLOCK_SIZE));
    hipFree(d_output);
    
    free(h_output);
    return result;
}
// ###### Reduction END #######

// ###### Reduction boolean count BEGIN #######

int reductionSequential_STOP_cryterion(bool *input, int width)
{
    bool sum = false;
    for (int i = 0; i < width; ++i)
    {
        sum = sum || input[i];
    }

    return sum;
}

__global__ void reductionKernelOp_STOP_cryterion(bool *input, bool *output, int width)
{
    //@@ INSERT CODE HERE
    __shared__ bool in[BLOCK_SIZE];
    if(threadIdx.x + (blockDim.x * blockIdx.x) < width){
        in[threadIdx.x] = !(input[threadIdx.x + (blockDim.x * blockIdx.x)]);
    }else{
        in[threadIdx.x] = false;
    }
    __syncthreads();
    int targetlevel = 0;
    int index = BLOCK_SIZE;
    while (index >>= 1) ++targetlevel;

    bool result = false;
    for(int it = 0; it < targetlevel; it++){
        if(threadIdx.x<(BLOCK_SIZE/(2<<it))){
            result = in[threadIdx.x] || in[threadIdx.x+(BLOCK_SIZE/(2<<it))];
            // printf("it: %d\t(%d, %d)\t%f\t%f\t%f\n",it,threadIdx.x, threadIdx.x+(1<<it) ,in[threadIdx.x] ,in[threadIdx.x+(1<<it)],result);
        }
        __syncthreads();
        if((threadIdx.x<(BLOCK_SIZE/(2<<it)))){
            in[threadIdx.x] = result;
        }
        //printf("%f\n",in[0]);
        __syncthreads();
    }
    
    if(threadIdx.x == 0){
        output[blockIdx.x] = in[0]; 
        // printf("%f",in[0]);
        // printf("\n%d\n",targetlevel);
    }
    
}

bool STOP_cryterion(bool* d_collision, int length)
{
    //@@ INSERT CODE HERE
    bool *d_output, *h_output;
    hipMalloc(&d_output, ceil((float)length/BLOCK_SIZE) * sizeof(bool));
    h_output = (bool*)malloc(ceil((float)length/BLOCK_SIZE) * sizeof(bool));

    dim3 dimGrid(ceil((float)length/BLOCK_SIZE),1,1);
    dim3 dimBlock(BLOCK_SIZE,1,1);
    
    reductionKernelOp_STOP_cryterion<<<dimGrid,dimBlock>>>(d_collision,d_output,length);
    
    hipMemcpy(h_output, d_output,  ceil((float)length/BLOCK_SIZE)*sizeof(bool), hipMemcpyDeviceToHost);

    bool result = reductionSequential_STOP_cryterion(h_output,ceil((float)length/BLOCK_SIZE));
    hipFree(d_output);
    
    free(h_output);
    return result;
}
// ###### Reduction END #######
// ###### Reduction MAX BEGIN #######

int reductionSequential_MAX(int *input, int width)
{
    int maximum = -1;
    for (int i = 0; i < width; ++i)
    {
        maximum = max(maximum,input[i]);
    }

    return maximum;
}

__global__ void reductionKernelOp_MAX(int *input, int *output, int width)
{
    //@@ INSERT CODE HERE
    __shared__ int in[BLOCK_SIZE];
    if(threadIdx.x + (blockDim.x * blockIdx.x) < width){
        in[threadIdx.x] = (input[threadIdx.x + (blockDim.x * blockIdx.x)]);
    }else{
        in[threadIdx.x] = -1;
    }
    __syncthreads();
    int targetlevel = 0;
    int index = BLOCK_SIZE;
    while (index >>= 1) ++targetlevel;

    int result = -1;
    for(int it = 0; it < targetlevel; it++){
        if(threadIdx.x<(BLOCK_SIZE/(2<<it))){
            result = (int)max((float)in[threadIdx.x],(float)in[threadIdx.x+(BLOCK_SIZE/(2<<it))]);
            // printf("it: %d\t(%d, %d)\t%d\t%d\n",it,threadIdx.x, threadIdx.x+(1<<it) ,in[threadIdx.x] ,in[threadIdx.x+(1<<it)]);
        }
        __syncthreads();
        if((threadIdx.x<(BLOCK_SIZE/(2<<it)))){
            in[threadIdx.x] = result;
        }
        //printf("%f\n",in[0]);
        __syncthreads();
    }
    
    if(threadIdx.x == 0){
        output[blockIdx.x] = in[0]; 
        // printf("%f",in[0]);
        // printf("\n%d\n",targetlevel);
    }
    
}

int MAX_cryterion(int* d_rewards, int length)
{
    //@@ INSERT CODE HERE
    int *d_output, *h_output;
    hipMalloc(&d_output, ceil((float)length/BLOCK_SIZE) * sizeof(int));
    h_output = (int*)malloc(ceil((float)length/BLOCK_SIZE) * sizeof(int));

    dim3 dimGrid(ceil((float)length/BLOCK_SIZE),1,1);
    dim3 dimBlock(BLOCK_SIZE,1,1);
    
    reductionKernelOp_MAX<<<dimGrid,dimBlock>>>(d_rewards,d_output,length);
    
    hipMemcpy(h_output, d_output,  ceil((float)length/BLOCK_SIZE)*sizeof(int), hipMemcpyDeviceToHost);

    int result = reductionSequential_MAX(h_output,ceil((float)length/BLOCK_SIZE));
    hipFree(d_output);
    
    free(h_output);
    return result;
}
// ###### Reduction MAX END #######


// ###### Reduction MAX IDX BEGIN #######

int reductionSequential_MAX_IDX(int *input, int *input_idx, int width)
{
    int maximum = -1;
    int idx = -1;
    for (int i = 0; i < width; ++i)
    {   
        if(maximum<input[i]){
            maximum = input[i];
            idx = input_idx[i];
        }
    }
    printf("Maximum: %d\n", maximum);
    return idx;
}

__global__ void reductionKernelOp_MAX_IDX(int *input, int *output, int *output_idx, int width)
{
    //@@ INSERT CODE HERE
    __shared__ int in[BLOCK_SIZE];
    __shared__ int in_idx[BLOCK_SIZE];
    if(threadIdx.x + (blockDim.x * blockIdx.x) < width){
        in[threadIdx.x] = (input[threadIdx.x + (blockDim.x * blockIdx.x)]);
        in_idx[threadIdx.x] = threadIdx.x + (blockDim.x * blockIdx.x);
    }else{
        in[threadIdx.x] = -1;
        in_idx[threadIdx.x] = -1;
    }
    __syncthreads();
    int targetlevel = 0;
    int index = BLOCK_SIZE;
    while (index >>= 1) ++targetlevel;

    int result = -1;
    int result_idx = -1;
    for(int it = 0; it < targetlevel; it++){
        if(threadIdx.x<(BLOCK_SIZE/(2<<it))){
            if(in[threadIdx.x]>in[threadIdx.x+(BLOCK_SIZE/(2<<it))]){
                result = in[threadIdx.x];
                result_idx = in_idx[threadIdx.x];
            }else{
                result = in[threadIdx.x+(BLOCK_SIZE/(2<<it))];
                result_idx = in_idx[threadIdx.x+(BLOCK_SIZE/(2<<it))];
            }
            
            // printf("it: %d\t(%d, %d)\t%d\t%d\n",it,threadIdx.x, threadIdx.x+(1<<it) ,in[threadIdx.x] ,in[threadIdx.x+(1<<it)]);
        }
        __syncthreads();
        if((threadIdx.x<(BLOCK_SIZE/(2<<it)))){
            in[threadIdx.x] = result;
            in_idx[threadIdx.x] = result_idx;
        }
        //printf("%f\n",in[0]);
        __syncthreads();
    }
    
    if(threadIdx.x == 0){
        output[blockIdx.x] = in[0]; 
        output_idx[blockIdx.x] = in_idx[0];
        // printf("%f",in[0]);
        // printf("\n%d\n",targetlevel);
    }
    
}

int MAX_cryterion_IDX(int* d_rewards, int length)
{
    //@@ INSERT CODE HERE
    int *d_output, *h_output,*d_output_idx, *h_output_idx;
    hipMalloc(&d_output, ceil((float)length/BLOCK_SIZE) * sizeof(int));
    hipMalloc(&d_output_idx, ceil((float)length/BLOCK_SIZE) * sizeof(int));
    h_output = (int*)malloc(ceil((float)length/BLOCK_SIZE) * sizeof(int));
    h_output_idx = (int*)malloc(ceil((float)length/BLOCK_SIZE) * sizeof(int));

    dim3 dimGrid(ceil((float)length/BLOCK_SIZE),1,1);
    dim3 dimBlock(BLOCK_SIZE,1,1);
    
    reductionKernelOp_MAX_IDX<<<dimGrid,dimBlock>>>(d_rewards,d_output,d_output_idx,length);
    
    hipMemcpy(h_output, d_output,  ceil((float)length/BLOCK_SIZE)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_output_idx, d_output_idx,  ceil((float)length/BLOCK_SIZE)*sizeof(int), hipMemcpyDeviceToHost);

    int result = reductionSequential_MAX_IDX(h_output,h_output_idx,ceil((float)length/BLOCK_SIZE));
    hipFree(d_output);
    hipFree(d_output_idx);
    
    free(h_output);
    free(h_output_idx);
    return result;
}
// ###### Reduction MAX IDX END #######

// Mnożenie macierzy CSR przez wektor 
__global__ void SparseMUL(int* column_idx, int* row_pointers, float* weights, float* input_vector, int vector_size, float* output_vector, int output_vector_size){
    int row = threadIdx.x + (blockDim.x * blockIdx.x);
    while(row<output_vector_size){
        float acc = 0;
        for(int col_idx=row_pointers[row]; col_idx<row_pointers[row+1]; col_idx++){
            acc += input_vector[column_idx[col_idx]] * weights[col_idx];
        }
        output_vector[row] = 1/(1+exp(-acc)); // sigmoida
        
        row += blockDim.x * gridDim.x;
    }
}

__global__ void updateRowPointers(int *row_pointers, int *out, int *blocks_edges, int *blocks_nodes, int no_instances, bool* enable){
    int inst = threadIdx.x + (blockDim.x * blockIdx.x);
    while(inst<no_instances){
        int idx = inst; // granulacja na poziomie idx per kernel
        for(int i=blocks_edges[idx]; i<blocks_edges[idx+1]; i++){
            if (enable[i])
                row_pointers[(out[i])+blocks_nodes[idx]+1] +=1;
        }
        inst += blockDim.x * gridDim.x;
    }
    
}

__global__ void updateCol_idx_weights(int *row_pointers_t, float *weights, int *col_idx, int *in, float *w, int *out, int *blocks_edges, int *blocks_nodes, int no_instances, bool* enable){
    int inst = threadIdx.x + (blockDim.x * blockIdx.x);
    while(inst<no_instances){
        int idx = inst;
        for(int i=blocks_edges[inst]; i<blocks_edges[inst+1]; i++){
            if (enable[i]){
                int temp = row_pointers_t[out[i] +blocks_nodes[idx]];
                row_pointers_t[out[i]+blocks_nodes[idx]] +=1; // no need for atomic add because [blocks_nodes[idx], blocks_nodes[idx]+1] is only for idx
                col_idx[temp] = in[i]+blocks_nodes[idx]; // jeżeli będą podwójne to nie ma znaczenia przy mnożeniu macierzowym
                weights[temp] = w[i];
            }
        }
        inst += blockDim.x * gridDim.x;
    }
}



class Pipe{
private:
    int y_upper_pipe;
    int y_bottom_pipe;
    int x_pipe;
    int r_pipe;
    int vel;

public:
    Pipe(int y_upper_pipe_, int y_bottom_pipe_, int x_pipe_, int r_pipe_, int vel_) : y_bottom_pipe(y_bottom_pipe_), y_upper_pipe(y_upper_pipe_), x_pipe(x_pipe_), r_pipe(r_pipe_), vel(vel_) {};

    Pipe(const Pipe& other) 
        : y_upper_pipe(other.y_upper_pipe), y_bottom_pipe(other.y_bottom_pipe),
          x_pipe(other.x_pipe), r_pipe(other.r_pipe), vel(other.vel) {
    }
    bool update(){
        x_pipe -= vel;
        if(x_pipe > -10) return 0;
        return 1;
    };
    int getX() const { return x_pipe; }
    int getYUpper() const { return y_upper_pipe; }
    int getYBottom() const { return y_bottom_pipe; }
    int getR() const { return r_pipe; }
    int getVelocity() const { return vel; }
    ~Pipe() = default;
};


__global__ void update_colisions(bool* colision, int* rewards, float* y, float x, int y_upper_pipe, int y_bottom_pipe, int x_pipe, int r_pipe, int no_instances){
    int i = threadIdx.x + (blockDim.x * blockIdx.x);
    while(i<no_instances){
        // printf("%f\n",y[i]);
        if((((x + FLOPPY_RADIUS >=(x_pipe-r_pipe)) && (x-FLOPPY_RADIUS)<=x_pipe+r_pipe) && (y[i]-FLOPPY_RADIUS<=y_upper_pipe || y[i]+FLOPPY_RADIUS>=HEIGHT-y_bottom_pipe)) || (y[i]>=HEIGHT-FLOPPY_RADIUS || y[i]<=0+FLOPPY_RADIUS)){
            colision[i] = true;
        }
        else{
            if(!colision[i]){
                rewards[i]++;
                
            }
        }
        i += blockDim.x * gridDim.x;
    }
}

__global__ void update_step(float* vel_y, float* y, float* vect_out,int *blocks_nodes, int no_instances){
    int i = threadIdx.x + (blockDim.x * blockIdx.x);
    while(i<no_instances){
        float g = 0.1;
        float flop_v = 4;
        vel_y[i] += g;
        if(vect_out[blocks_nodes[i]]>0.5){
            vel_y[i] = -flop_v;
        }
        y[i]+=vel_y[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void update_instance_inputs(int *blocks_nodes, float* vect_in, int y_upper_pipe, int y_bottom_pipe, int x_pipe, float x, float* y, int no_instances){
    int i = threadIdx.x + (blockDim.x * blockIdx.x); // zrównoleglenie na instancje
    while(i<no_instances){
        float x1 = (x_pipe - x)/400.0;
        float x2 = (y_bottom_pipe - y[i])/600.0;
        float x3 = (y[i] - y_upper_pipe)/600.0;
        vect_in[blocks_nodes[i] + 1] = x1;
        vect_in[blocks_nodes[i] + 2] = x2;
        vect_in[blocks_nodes[i] + 3] = x3;
        // printf("%d\t%d\t%f\t%f\t%f\n",i, blocks_nodes[i],vect_in[blocks_nodes[i] + 1],vect_in[blocks_nodes[i] + 2],vect_in[blocks_nodes[i] + 3]);
        i += blockDim.x * gridDim.x;
    }
    
}

__global__ void initialize(float* array, float value, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        array[idx] = value;
    }
}


void symulate(int *blocks_nodes, int* column_idx, int* row_pointers, float* weights, int* rewards, int vector_size, int no_instances, int max_iterations, int no_nodes, int no_edges){
    // inicjalizacja
    Pipe now = Pipe(300,HEIGHT-(300 + 2*GAP_R),440,40,1);
    Pipe next = Pipe(120,HEIGHT-(120 + 2*GAP_R),880,40,1);
    Pipe prev = Pipe(300,HEIGHT-(300 + 2*GAP_R),440,40,1);
    // bool* isFlopping;
    // isFlopping = (bool*) malloc((no_instances) * sizeof(bool));
    float x = 120; // takie same x dla populaci
    float* d_y;
    dim3 dimGrid(ceil((float)(no_instances)/BLOCK_SIZE),1,1);
    dim3 dimBlock(BLOCK_SIZE,1,1);
    hipMalloc(&d_y, (no_instances) * sizeof(float));
    initialize<<<dimGrid, dimBlock>>>(d_y, 50.0f, no_instances);

    float* d_vel_y;
    hipMalloc(&d_vel_y, (no_instances) * sizeof(float));
    hipMemset(d_vel_y, 0.0, (no_instances) * sizeof(float));

    bool* d_colision;// inicialize to false
    hipMalloc(&d_colision, (no_instances) * sizeof(bool));
    hipMemset(d_colision, false, (no_instances) * sizeof(bool));

    // internal state vectors:
    float* d_vect_in;
    float* d_vect_out;
    float* d_vect_temp;
    

    hipMalloc(&d_vect_in, (no_nodes) * sizeof(float));
    hipMemset(d_vect_in, 0.0, (no_nodes) * sizeof(float));
    hipMalloc(&d_vect_out, (no_nodes) * sizeof(float));
    hipMemset(d_vect_out, 0.0, (no_nodes) * sizeof(float));
    // koniec inicjalizacji
    
    for(int it = 0; it < max_iterations; it++){ // główna pętla
        
        update_colisions<<<dimGrid,dimBlock>>>(d_colision, rewards, d_y, x, now.getYUpper(), now.getYBottom(), now.getX(), now.getR(), no_instances);
        bool STOP = STOP_cryterion(d_colision, no_instances);
        // printf("%d\t%d\n", STOP, it);
        if(!STOP) break;
        // #### UPDATE ENV ####
        if(x - FLOPPY_RADIUS > now.getX() + now.getR()){
            prev = now;
            now = next;
            int randomNum = rand() %(HEIGHT - 2*GAP_R) + GAP_R;
            next = Pipe(randomNum - GAP_R,HEIGHT - (randomNum + GAP_R),880,40,1);
        }
        // #### UPDATE ENV END ####
        // #### Control ####
        d_vect_temp = d_vect_in;
        d_vect_in = d_vect_out;
        d_vect_out = d_vect_temp; // swap important as not to allocate memory (everything on device side)
        hipMemset(d_vect_out, 0.0, (no_nodes) * sizeof(float));

        update_instance_inputs<<<dimGrid,dimBlock>>>(blocks_nodes, d_vect_in, now.getYUpper(), now.getYBottom(), now.getX(), x, d_y, no_instances);

        dim3 dimGrid2(ceil((float)(no_nodes)/BLOCK_SIZE),1,1);
        dim3 dimBlock2(BLOCK_SIZE,1,1);
        SparseMUL<<<dimGrid2, dimBlock2>>>(column_idx, row_pointers, weights, d_vect_in, no_nodes, d_vect_out, no_nodes);

        // #### update step ####
        update_step<<<dimGrid,dimBlock>>>(d_vel_y, d_y, d_vect_out, blocks_nodes, no_instances);

        prev.update();
        now.update();
        next.update();
    }
    hipFree(d_y);
    hipFree(d_vel_y);
    hipFree(d_colision);
    hipFree(d_vect_in);
    hipFree(d_vect_out);
}

void symulate(
    int *d_in,
    int *d_out,
    float *d_w,
    bool *d_enabled,
    int *d_innov,
    int *d_blocks_edges,
    int *d_translation,
    int *d_blocks_nodes,
    int* d_rewards,
    int no_instances,
    int no_nodes,
    int no_edges,
    int max_it
){
    // ##### PREPARATION TO SYMULATION #####
    // TO CSR MATRIX
    int colsize = count_col_size(d_enabled, no_edges); // już zrównoleglone za pomocą redukcji

    int *d_col_idx; // size w
    float *d_weights; // size w
    int *d_row_pointers; //size translation+1
    
    hipMalloc(&d_col_idx, (colsize) * sizeof(int));
    hipMalloc(&d_weights, (colsize) * sizeof(float));
    hipMalloc(&d_row_pointers, (no_nodes + 1) * sizeof(int));

    hipMemset(d_row_pointers, 0, (no_nodes + 1) * sizeof(int));

    dim3 dimGrid(ceil((float)(no_instances)/BLOCK_SIZE),1,1);
    dim3 dimBlock(BLOCK_SIZE,1,1);

    updateRowPointers<<<dimGrid,dimBlock>>>(d_row_pointers, d_out, d_blocks_edges, d_blocks_nodes, no_instances, d_enabled);

    

    int *d_row_pointers_t;
    hipMalloc(&d_row_pointers_t, (no_nodes + 1) * sizeof(int));

    cumulatedHistogram_with_copy(d_row_pointers, d_row_pointers_t, d_row_pointers, (no_nodes + 1));

    

    updateCol_idx_weights<<<dimGrid,dimBlock>>>(d_row_pointers_t, d_weights, d_col_idx, d_in, d_w, d_out, d_blocks_edges, d_blocks_nodes, no_instances, d_enabled);
    hipFree(d_row_pointers_t); // important free
    
    // int* h_array = (int*)malloc((colsize) * sizeof(int));
    // hipMemcpy(h_array, d_col_idx, (colsize) * sizeof(int), hipMemcpyDeviceToHost);
    // for(int i=0; i<(colsize); i++) printf("%d\t", h_array[i]);
    // printf("\n");

    hipMemset(d_rewards, -1, (no_instances) * sizeof(int));//set to -1

    
    // symulate
    symulate(d_blocks_nodes, d_col_idx, d_row_pointers, d_weights, d_rewards, no_nodes, no_instances, max_it, no_nodes, no_edges); // max iteration to 10
}

__global__ void scanKernelX(int *Y, int *S, int *X, int width)
{
    //@@ INSERT CODE HERE
    __shared__ int in[SECTION_SIZE];
    if(threadIdx.x + (blockDim.x * blockIdx.x) < width){
        in[threadIdx.x] = X[threadIdx.x + (blockDim.x * blockIdx.x)];
    }else{
        in[threadIdx.x] = 0;
    }
    __syncthreads();
    
    int result = 0;
    for(int start = 1; ((start + (blockDim.x * blockIdx.x)) < width) && (start < blockDim.x); start=start*2){
        if(threadIdx.x>=start){
            result = in[threadIdx.x] + in[threadIdx.x-start];
            // printf("it: %d\t(%d, %d)\t%f\t%f\t%f\n",it,threadIdx.x, threadIdx.x+(1<<it) ,in[threadIdx.x] ,in[threadIdx.x+(1<<it)],result);
        }
        __syncthreads();
        if(threadIdx.x>=start){
            in[threadIdx.x] = result;
        }
        //printf("%f\n",in[0]);
        __syncthreads();
    }
    
    if(threadIdx.x + (blockDim.x * blockIdx.x) < width){
        Y[threadIdx.x + (blockDim.x * blockIdx.x)] = in[threadIdx.x]; 
        if(threadIdx.x == (blockDim.x - 1)){
            S[blockIdx.x] = in[threadIdx.x];
        }
    }
    
}

__global__ void scanKernelS(int *S, int width)
{
    //@@ INSERT CODE HERE
    __shared__ int in[SECTION_SIZE];
    if(threadIdx.x + (blockDim.x * blockIdx.x) < width){
        in[threadIdx.x] = S[threadIdx.x + (blockDim.x * blockIdx.x)];
    }else{
        in[threadIdx.x] = 0;
    }
    __syncthreads();

    int result = 0;
    for(int start = 1; start < width; start=start*2){
        if(threadIdx.x>=start){
            result = in[threadIdx.x] + in[threadIdx.x-start];
            // printf("it: %d\t(%d, %d)\t%f\t%f\t%f\n",it,threadIdx.x, threadIdx.x+(1<<it) ,in[threadIdx.x] ,in[threadIdx.x+(1<<it)],result);
        }
        __syncthreads();
        if(threadIdx.x>=start){
            in[threadIdx.x] = result;
        }
        //printf("%f\n",in[0]);
        __syncthreads();
    }
    
    if(threadIdx.x < width){
        S[threadIdx.x] = in[threadIdx.x]; 
    }
}

__global__ void updateYKernel(int *Y, int *S, int widthY)
{
    //@@ INSERT CODE HERE
    if((blockIdx.x >= 1) && (threadIdx.x + (blockDim.x * blockIdx.x) < widthY)){
        Y[threadIdx.x + (blockDim.x * blockIdx.x)] += S[blockIdx.x - 1];
    }
}

void cumulatedHistogram(int *d_Y, int *d_X, int width)
{
    /*
    input:
        device vectors:
            d_X - input
            d_Y - output
        width - size of vectors
    */
    int *d_S;
    hipMalloc(&d_S, ceil((float)width/SECTION_SIZE) * sizeof(int));

    dim3 dimGrid(ceil((float)width/SECTION_SIZE),1,1);
    dim3 dimBlock(SECTION_SIZE,1,1);
    
    scanKernelX<<<dimGrid,dimBlock>>>(d_Y, d_S, d_X, width);

    dim3 dimGrid2(1,1,1);
    dim3 dimBlock2(SECTION_SIZE,1,1);

    scanKernelS<<<dimGrid2,dimBlock2>>>(d_S, ceil((float)width/SECTION_SIZE));
    
    updateYKernel<<<dimGrid,dimBlock>>>(d_Y, d_S, width);

    hipFree(d_S);
}
// ###### CUMULATED HISTOGRAM END #######

__global__ void countOffsprings(int *first_pair, int *second_pair, int no_offsprings, int *innov, int *blocks_edges, int no_instances, int *length_offspring, int offset)
{
    int i = threadIdx.x + (blockDim.x * blockIdx.x);
    while(i<no_offsprings){
        int first = first_pair[i];
        int second = second_pair[i];
        int idx_first = blocks_edges[first];
        int idx_second = blocks_edges[second];
        int acc = 0;
        // if(i == 0) printf("\nstart\n");
        while(idx_first != blocks_edges[first+1] && idx_second != blocks_edges[second+1]){
            // if(i == 0) printf("\ninnov: %d, %d\t %d, %d\n", innov[idx_first],innov[idx_second], idx_first, idx_second);
            if(innov[idx_first] == innov[idx_second]){
                // if(i == 0) printf("%d", innov[idx_first]);
                idx_first++;
                idx_second++;
            }else if(innov[idx_first] < innov[idx_second]){
                // if(i == 0) printf("%d\t", innov[idx_first]);
                idx_first++;
            }else{
                // if(i == 0) printf("%d\t", innov[idx_second]);
                idx_second++;
            }
            acc++;
        }
        // if(i == 0) printf("\tend\t");
        while(idx_first != blocks_edges[first+1]){
            // if(i == 0) printf("%d\t", innov[idx_second]);
            idx_first++;
            acc++;
        }
        while(idx_second != blocks_edges[second+1]){
            // if(i == 0) printf("%d\t", innov[idx_second]);
            idx_second++;
            acc++;
        }
        length_offspring[i+offset]=acc;
        // if(i == 0) printf("\n%d\n", length_offspring[i+offset]);
        // if(i == 0) printf("\nendend\n");
        i += blockDim.x * gridDim.x;
    }
}

__global__ void countOffspringsNodes(int *first_pair, int *second_pair, int no_offsprings, int *translation, int *blocks_nodes, int no_instances, int *length_offspringNodes, int offset){
    int i = threadIdx.x + (blockDim.x * blockIdx.x);
    while(i<no_offsprings){
        int first = first_pair[i];
        int second = second_pair[i];
        int idx_first = blocks_nodes[first];
        int idx_second = blocks_nodes[second];
        int acc = 0;
        while(idx_first != blocks_nodes[first+1] && idx_second != blocks_nodes[second+1]){
            if(translation[idx_first] == translation[idx_second]){
                idx_first++;
                idx_second++;
            }else if(translation[idx_first] < translation[idx_second]){
                idx_first++;
            }else{
                idx_second++;
            }
            acc++;
        }
        while(idx_first != blocks_nodes[first+1]){
            idx_first++;
            acc++;
        }
        while(idx_second != blocks_nodes[second+1]){
            idx_second++;
            acc++;
        }
        length_offspringNodes[i+offset]=acc;

        i += blockDim.x * gridDim.x;
    }
}

__global__ void countSurvivors(int *blocks_edges, int *blocks_nodes, int *new_blocks_edges, int *new_blocks_nodes, int no_survivors, int *no_instance_seq){
    int i = threadIdx.x + (blockDim.x * blockIdx.x);
    while(i<no_survivors){
        int instance_number = no_instance_seq[i];
        // printf("%d\t%d\t%d\t%d\n", instance_number, 1+i, blocks_edges[instance_number+1]-blocks_edges[instance_number], instance_number);
        new_blocks_nodes[1+i] = blocks_nodes[instance_number+1]-blocks_nodes[instance_number];
        new_blocks_edges[1+i] = blocks_edges[instance_number+1]-blocks_edges[instance_number];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void copySurvivors(int *istance_numbers_seq, int *blocks_edges, int *blocks_nodes, int *new_blocks_nodes, int *new_blocks_edges, int no_survivors,
    int *new_in, int *new_out, float *new_w, bool *new_enabled, int *new_innov, int *new_translation, 
    int *in, int *out, float *w, bool *enabled, int *innov, int *translation){

    int i = threadIdx.x + (blockDim.x * blockIdx.x);
    while(i<no_survivors){
        // 
        for(int j=0; j<new_blocks_edges[i+1] - new_blocks_edges[i]; j++){
            new_in[j+new_blocks_edges[i]] = in[j + blocks_edges[istance_numbers_seq[i]]];
            new_out[j+new_blocks_edges[i]] = out[j + blocks_edges[istance_numbers_seq[i]]];
            new_enabled[j+new_blocks_edges[i]] = enabled[j + blocks_edges[istance_numbers_seq[i]]];
            new_innov[j+new_blocks_edges[i]] = innov[j + blocks_edges[istance_numbers_seq[i]]];
            new_w[j+new_blocks_edges[i]] = w[j + blocks_edges[istance_numbers_seq[i]]];
        }
        for(int j=0; j<new_blocks_nodes[i+1] - new_blocks_nodes[i]; j++){
            new_translation[j+new_blocks_nodes[i]] = translation[j + blocks_nodes[istance_numbers_seq[i]]];
        }
        i += blockDim.x * gridDim.x;
    }
}

__global__ void crossover(
    int *blocks_nodes,
    int *blocks_edges,
    int *new_blocks_nodes,
    int *new_blocks_edges,
    int offset,
    int *in,
    int *out,
    float *w,
    bool *enabled,
    int *innov,
    int *first_pair,
    int *second_pair,
    int no_offsprings,
    int *new_in,
    int *new_out,
    float *new_w,
    bool *new_enabled,
    int *new_innov,
    int *new_translation,
    int *translation,
    int *translation_t1,
    int *translation_t2
    ){
    int i = threadIdx.x + (blockDim.x * blockIdx.x);
    while(i<no_offsprings){
        int first = first_pair[i];
        int second = second_pair[i];
        int idx_first = blocks_nodes[first];
        int idx_second = blocks_nodes[second];
        int idx = 0;
        // nodes translations
        while(idx_first != blocks_nodes[first+1] && idx_second != blocks_nodes[second+1]){
            if(translation[idx_first] == translation[idx_second]){
                new_translation[idx + new_blocks_nodes[i+offset]] = translation[idx_first];

                translation_t1[idx_first - blocks_nodes[first] + new_blocks_nodes[i+offset]] = idx;
                translation_t2[idx_second - blocks_nodes[second] + new_blocks_nodes[i+offset]] = idx;
                idx++;
                idx_first++;
                idx_second++;
            }else if(translation[idx_first] < translation[idx_second]){
                new_translation[idx + new_blocks_nodes[i+offset]] = translation[idx_first];
                translation_t1[idx_first - blocks_nodes[first] + new_blocks_nodes[i+offset]] = idx;
                idx++;
                idx_first++;
            }else{
                new_translation[idx + new_blocks_nodes[i+offset]] = translation[idx_second];
                translation_t2[idx_second - blocks_nodes[second] + new_blocks_nodes[i+offset]] = idx;
                idx++;
                idx_second++;
            }
        }
        while(idx_first != blocks_nodes[first+1]){
            new_translation[idx + new_blocks_nodes[i+offset]] = translation[idx_first];
            translation_t1[idx_first - blocks_nodes[first] + new_blocks_nodes[i+offset]] = idx;
            idx++;
            idx_first++;
        }
        while(idx_second != blocks_nodes[second+1]){
            new_translation[idx + new_blocks_nodes[i+offset]] = translation[idx_second];
            translation_t2[idx_second - blocks_nodes[second] + new_blocks_nodes[i+offset]] = idx;
            idx++;
            idx_second++;
        }

        // edges
        idx_first = blocks_edges[first];
        idx_second = blocks_edges[second];
        idx = 0;
        // if(i==0) printf("\nSTART\n");
        while(idx_first != blocks_edges[first+1] && idx_second != blocks_edges[second+1]){
            
            if(innov[idx_first] == innov[idx_second]){ // innov - only innovation number differs edges
                new_w[idx+new_blocks_edges[i+offset]] = (w[idx_first] + w[idx_second])/2;
                new_enabled[idx+new_blocks_edges[i+offset]] = enabled[idx_first] && enabled[idx_second];
                new_innov[idx+new_blocks_edges[i+offset]] = innov[idx_first];
                // if(i==0) printf("%d, %d\t", in[idx_first]+blocks_nodes[first_pair[i]],translation_t[in[idx_first]]+blocks_nodes[first_pair[i]]);
                new_in[idx+new_blocks_edges[i+offset]] = translation_t1[in[idx_first]+ new_blocks_nodes[i+offset]];
                new_out[idx+new_blocks_edges[i+offset]] = translation_t1[out[idx_first]+ new_blocks_nodes[i+offset]];
                if(new_in[idx+new_blocks_edges[i+offset]]<0) printf("KRZYZ1\t%d\t%d\t%d\n",idx+new_blocks_edges[i+offset], i+offset, translation_t1[in[idx_first]+ new_blocks_nodes[i+offset]]);
                idx_first++;
                idx_second++;
                idx++;
            }else if(innov[idx_first] < innov[idx_second]){
                // if(i==0) printf("%d, %d\t", in[idx_first]+blocks_nodes[first_pair[i]],translation_t[in[idx_first]+blocks_nodes[first_pair[i]]]);
                new_w[idx+new_blocks_edges[i+offset]] = w[idx_first];
                new_enabled[idx+new_blocks_edges[i+offset]] = enabled[idx_first];
                new_innov[idx+new_blocks_edges[i+offset]] = innov[idx_first];
                new_in[idx+new_blocks_edges[i+offset]] = translation_t1[in[idx_first]+ new_blocks_nodes[i+offset]];
                if(new_in[idx+new_blocks_edges[i+offset]]<0) printf("KRZYZ1\t%d\t%d\t%d\n",idx+new_blocks_edges[i+offset], i+offset, translation_t1[in[idx_first]+ new_blocks_nodes[i+offset]]);
                new_out[idx+new_blocks_edges[i+offset]] = translation_t1[out[idx_first]+ new_blocks_nodes[i+offset]];
                idx_first++;
                idx++;
            }else{
                // if(i==0) printf("%d, %d\t", in[idx_second]+blocks_nodes[second_pair[i]],translation_t[in[idx_second]+blocks_nodes[second_pair[i]]]);
                new_w[idx+new_blocks_edges[i+offset]] = w[idx_second];
                new_enabled[idx+new_blocks_edges[i+offset]] = enabled[idx_second];
                new_innov[idx+new_blocks_edges[i+offset]] = innov[idx_second];
                new_in[idx+new_blocks_edges[i+offset]] = translation_t2[in[idx_second]+ new_blocks_nodes[i+offset]];
                if(new_in[idx+new_blocks_edges[i+offset]]<0) printf("KRZYZ1\t%d\t%d\t%d\n",idx+new_blocks_edges[i+offset], i+offset, translation_t2[in[idx_second]+ new_blocks_nodes[i+offset]]);
                new_out[idx+new_blocks_edges[i+offset]] = translation_t2[out[idx_second]+ new_blocks_nodes[i+offset]];
                idx_second++;
                idx++;
            }

        }
        while(idx_first != blocks_edges[first+1]){
            // if(i==0) printf("%d, %d\t", in[idx_first]+blocks_nodes[first_pair[i]],translation_t[in[idx_first]+blocks_nodes[first_pair[i]]]);
            new_w[idx+new_blocks_edges[i+offset]] = w[idx_first];
            new_enabled[idx+new_blocks_edges[i+offset]] = enabled[idx_first];
            new_innov[idx+new_blocks_edges[i+offset]] = innov[idx_first];
            new_in[idx+new_blocks_edges[i+offset]] = translation_t1[in[idx_first]+ new_blocks_nodes[i+offset]];
            if(new_in[idx+new_blocks_edges[i+offset]]<0) printf("KRZYZ1\t%d\t%d\t%d\n",idx+new_blocks_edges[i+offset], i+offset, translation_t1[in[idx_first]+ new_blocks_nodes[i+offset]]);
            new_out[idx+new_blocks_edges[i+offset]] = translation_t1[out[idx_first]+ new_blocks_nodes[i+offset]];
            idx_first++;
            idx++;
        }
        while(idx_second != blocks_edges[second+1]){
            // if(i==0) printf("%d, %d\t", in[idx_second]+blocks_nodes[second_pair[i]],translation_t[in[idx_second]+blocks_nodes[second_pair[i]]]);
            new_w[idx+new_blocks_edges[i+offset]] = w[idx_second];
            new_enabled[idx+new_blocks_edges[i+offset]] = enabled[idx_second];
            new_innov[idx+new_blocks_edges[i+offset]] = innov[idx_second];
            new_in[idx+new_blocks_edges[i+offset]] = translation_t2[in[idx_second]+ new_blocks_nodes[i+offset]];
            if(new_in[idx+new_blocks_edges[i+offset]]<0) printf("KRZYZ1\t%d\t%d\t%d\n",idx+new_blocks_edges[i+offset], i+offset, translation_t2[in[idx_second]+ new_blocks_nodes[i+offset]]);
            new_out[idx+new_blocks_edges[i+offset]] = translation_t2[out[idx_second]+ new_blocks_nodes[i+offset]];
            idx_second++;
            idx++;
        }
        // if(i==0) printf("\nEND\n");
        i += blockDim.x * gridDim.x;
    }
    
}
__global__ void initialize_rng(hiprandState* state, unsigned long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(clock64()+seed, idx, 0, &state[idx]);
}

__global__ void selection_step(hiprandState* state,int *rewards, int *mask, int no_instances){
    int i = threadIdx.x + (blockDim.x * blockIdx.x);
    while(i<SURVIVORS_TOURNAMENTS){
        int best_idx = 0;
        int best_reward = -1;
        for(int k=0; k<K; k++){
            int idx = (int)((1.0-hiprand_uniform(&state[i])) * no_instances);
            if(rewards[idx]>best_reward){
                best_reward = rewards[idx];
                best_idx = idx;
            }
        }
        mask[best_idx] = 1;
        i += blockDim.x * gridDim.x;
    }
}

__global__ void selection_crossover_step(hiprandState* state,int *rewards, int *first_parent, int *second_parent, int no_offspring, int no_instances){
    int i = threadIdx.x + (blockDim.x * blockIdx.x);
    while(i<no_offspring){
        int best_idx = 0;
        int best_reward = -1;
        for(int k=0; k<K; k++){
            int idx = (int)((1.0-hiprand_uniform(&state[i])) * no_instances);
            if(rewards[idx]>best_reward){
                best_reward = rewards[idx];
                best_idx = idx;
            }
        }
        first_parent[i] = best_idx;
        best_idx = 0;
        best_reward = -1;
        for(int k=0; k<K; k++){
            int idx = (int)((1.0-hiprand_uniform(&state[i])) * no_instances);
            if(rewards[idx]>best_reward){
                best_reward = rewards[idx];
                best_idx = idx;
            }
        }
        second_parent[i] = best_idx;
        i += blockDim.x * gridDim.x;
    }
}

__global__ void countMutations(hiprandState* state, int* mutation_parent, int* rewards ,int *blocks_edges, int* blocks_nodes, int *new_blocks_edges, int* new_blocks_nodes, int no_instances, int offset){
    int i = threadIdx.x + (blockDim.x * blockIdx.x);
    while(i<MUTATION_T1 + MUTATION_T2){
        int best_idx = 0;
        int best_reward = -1;
        for(int k=0; k<K; k++){
            int idx = (int)((1.0-hiprand_uniform(&state[i])) * no_instances);
            if(rewards[idx]>best_reward){
                best_reward = rewards[idx];
                best_idx = idx;
            }
        }
        if(i<MUTATION_T1){
            // printf("%d,%d\n", offset,i+offset);
            new_blocks_edges[i+offset+1] = blocks_edges[best_idx+1] - blocks_edges[best_idx] + 2;
            new_blocks_nodes[i+offset+1] = blocks_nodes[best_idx+1] - blocks_nodes[best_idx] + 1;
            mutation_parent[i] = best_idx;
        }else{
            // printf("%d,%d\n", offset,i+offset);
            new_blocks_edges[i+offset+1] = blocks_edges[best_idx+1] - blocks_edges[best_idx] + 1;
            new_blocks_nodes[i+offset+1] = blocks_nodes[best_idx+1] - blocks_nodes[best_idx];
            mutation_parent[i] = best_idx;
        }

        i += blockDim.x * gridDim.x;
    }
}

__global__ void MutateT1T2(
    hiprandState* state,
    int* mutation_parent,
    int *blocks_nodes,
    int *blocks_edges,
    int *new_blocks_nodes,
    int *new_blocks_edges,
    int offset,
    int *in,
    int *out,
    float *w,
    bool *enabled,
    int *innov,
    int *new_in,
    int *new_out,
    float *new_w,
    bool *new_enabled,
    int *new_innov,
    int *new_translation,
    int *translation,
    int offset_edges_innov,
    int offset_nodes_innov
    ){
    int i = threadIdx.x + (blockDim.x * blockIdx.x);
    while(i<MUTATION_T1 + MUTATION_T2){
        if(i<MUTATION_T1){
            // chose random edge:
            int length_edges = blocks_edges[mutation_parent[i]+1] - blocks_edges[mutation_parent[i]];
            int split_idx = (int)((1.0-hiprand_uniform(&state[i])) * length_edges);

            for(int j=0; j<new_blocks_edges[offset+i+1] - new_blocks_edges[offset+i] - 2; j++){
                new_in[j+new_blocks_edges[offset+i]] = in[j + blocks_edges[mutation_parent[i]]];
                if(in[j + blocks_edges[mutation_parent[i]]]<0) printf("MUTACJA\t");
                new_out[j+new_blocks_edges[offset+i]] = out[j + blocks_edges[mutation_parent[i]]];
                if(split_idx != j){
                    new_enabled[j+new_blocks_edges[offset+i]] = enabled[j + blocks_edges[mutation_parent[i]]];
                }else{
                    new_enabled[j+new_blocks_edges[offset+i]] = false;
                }
                new_innov[j+new_blocks_edges[offset+i]] = innov[j + blocks_edges[mutation_parent[i]]];
                new_w[j+new_blocks_edges[offset+i]] = w[j + blocks_edges[mutation_parent[i]]];
            }
            // new edges
            new_in[new_blocks_edges[offset+i+1] - new_blocks_edges[offset+i] - 2 +new_blocks_edges[offset+i]] = in[split_idx+blocks_edges[mutation_parent[i]]];
            new_out[new_blocks_edges[offset+i+1] - new_blocks_edges[offset+i] - 2 +new_blocks_edges[offset+i]] = new_blocks_nodes[offset+i+1] - new_blocks_nodes[offset+i] - 1;
            new_enabled[new_blocks_edges[offset+i+1] - new_blocks_edges[offset+i] - 2 +new_blocks_edges[offset+i]] = true;
            new_innov[new_blocks_edges[offset+i+1] - new_blocks_edges[offset+i] - 2 +new_blocks_edges[offset+i]] = offset_edges_innov + 2*i;
            new_w[new_blocks_edges[offset+i+1] - new_blocks_edges[offset+i] - 2 +new_blocks_edges[offset+i]] = w[split_idx+blocks_edges[mutation_parent[i]]];
            if(in[split_idx+blocks_edges[mutation_parent[i]]]<0) printf("MUTACJA\t");
            new_in[new_blocks_edges[offset+i+1] - new_blocks_edges[offset+i] - 1 +new_blocks_edges[offset+i]] = new_blocks_nodes[offset+i+1] - new_blocks_nodes[offset+i] - 1;
            new_out[new_blocks_edges[offset+i+1] - new_blocks_edges[offset+i] - 1 +new_blocks_edges[offset+i]] = out[split_idx+blocks_edges[mutation_parent[i]]];
            new_enabled[new_blocks_edges[offset+i+1] - new_blocks_edges[offset+i] - 1 +new_blocks_edges[offset+i]] = true;
            new_innov[new_blocks_edges[offset+i+1] - new_blocks_edges[offset+i] - 1 +new_blocks_edges[offset+i]] = offset_edges_innov + 2*i + 1;
            new_w[new_blocks_edges[offset+i+1] - new_blocks_edges[offset+i] - 1 +new_blocks_edges[offset+i]] = w[split_idx+blocks_edges[mutation_parent[i]]];

            for(int j=0; j<new_blocks_nodes[offset+i+1] - new_blocks_nodes[offset+i] - 1; j++){
                // printf("\n%d, %d\n", j+new_blocks_nodes[offset+i], translation[j + blocks_nodes[mutation_parent[i]]]);
                new_translation[j+new_blocks_nodes[offset+i]] = translation[j + blocks_nodes[mutation_parent[i]]];
            }
            // new nodes
            new_translation[new_blocks_nodes[offset+i] + new_blocks_nodes[offset+i+1] - new_blocks_nodes[offset+i] - 1] = offset_nodes_innov + i;

        }else{
            int length_nodes = new_blocks_nodes[offset+i+1] - new_blocks_nodes[offset+i];
            int first_idx = (int)((1.0-hiprand_uniform(&state[i])) * length_nodes);
            int second_idx = (int)((1.0-hiprand_uniform(&state[i])) * length_nodes);

            for(int j=0; j<new_blocks_edges[offset+i+1] - new_blocks_edges[offset+i] - 1; j++){
                // printf("%d\n", j+new_blocks_edges[offset+i+1]);
                new_in[j+new_blocks_edges[offset+i]] = in[j + blocks_edges[mutation_parent[i]]];
                new_out[j+new_blocks_edges[offset+i]] = out[j + blocks_edges[mutation_parent[i]]];
                new_enabled[j+new_blocks_edges[offset+i]] = enabled[j + blocks_edges[mutation_parent[i]]];
                new_innov[j+new_blocks_edges[offset+i]] = innov[j + blocks_edges[mutation_parent[i]]];
                new_w[j+new_blocks_edges[offset+i]] = w[j + blocks_edges[mutation_parent[i]]];
            }

            new_in[new_blocks_edges[offset+i+1] - new_blocks_edges[offset+i] - 1 +new_blocks_edges[offset+i]] = first_idx;
            new_out[new_blocks_edges[offset+i+1] - new_blocks_edges[offset+i] - 1 +new_blocks_edges[offset+i]] = second_idx;
            new_enabled[new_blocks_edges[offset+i+1] - new_blocks_edges[offset+i] - 1 +new_blocks_edges[offset+i]] = true;
            new_innov[new_blocks_edges[offset+i+1] - new_blocks_edges[offset+i] - 1 +new_blocks_edges[offset+i]] = offset_edges_innov + 2*MUTATION_T1 + (i - MUTATION_T1);
            new_w[new_blocks_edges[offset+i+1] - new_blocks_edges[offset+i] - 1 +new_blocks_edges[offset+i]] = (float)((1.0-hiprand_uniform(&state[i])) - 0.5);
            if(first_idx<0) printf("MUTACJA\t");
            for(int j=0; j<new_blocks_nodes[offset+i+1] - new_blocks_nodes[offset+i]; j++){
                new_translation[j+new_blocks_nodes[offset+i]] = translation[j + blocks_nodes[mutation_parent[i]]];
            }
        }
        i += blockDim.x * gridDim.x;
    }
}

__global__ void mutate_weights(hiprandState* state, float *w, int no_edges){
    int i = threadIdx.x + (blockDim.x * blockIdx.x);
    while(i<no_edges){
        if(hiprand_uniform(&state[i])<MUTATION_W_P){
            w[i] += hiprand_uniform(&state[i]) - 0.5;
        }
        i += blockDim.x * gridDim.x;
    }
}


void get_new_population(
    int **d_in_,
    int **d_out_,
    float **d_w_,
    bool **d_enabled_,
    int **d_innov_,
    int **d_blocks_edges_,
    int **d_translation_,
    int **d_blocks_nodes_,
    int *d_rewards,
    int no_instances,
    int *no_nodes_,
    int *no_edges_
    ){
    // przepisanie
    int *d_in;
    int *d_out;
    float *d_w;
    bool *d_enabled;
    int *d_innov;
    
    int *d_blocks_edges;
    int *d_translation;
    int *d_blocks_nodes;

    d_in = *d_in_;
    d_out = *d_out_;
    d_w = *d_w_;
    d_enabled = *d_enabled_;
    d_innov = *d_innov_;

    d_blocks_edges = *d_blocks_edges_;
    d_translation = *d_translation_;
    d_blocks_nodes = *d_blocks_nodes_;
    //


    int no_survivors; // number of 1 in mask
    int no_mutations; // TODO: mutacia

    int no_offsprings; // ze starej populacji

    int *d_mask;
    int *d_first_pair;
    int *d_second_pair;

    hiprandState *d_state;
    int NGrid = ceil((float)(SURVIVORS_TOURNAMENTS)/BLOCK_SIZE);

    hipMalloc(&d_mask, no_instances * sizeof(int));
    hipMalloc(&d_state, NGrid * BLOCK_SIZE * sizeof(hiprandState));

    initialize_rng<<<NGrid, BLOCK_SIZE>>>(d_state, time(0));
    selection_step<<<NGrid, BLOCK_SIZE>>>(d_state, d_rewards, d_mask, no_instances);
    hipFree(d_state);
    
    

    int *h_mask;
    h_mask = (int*) malloc(no_instances * sizeof(int));
    hipMemcpy(h_mask, d_mask, no_instances * sizeof(int), hipMemcpyDeviceToHost);
    int idx = 0;
    for(int i=0; i<no_instances; i++){ // to będzie sekwencyjne bo tylko przepisanie
        if(h_mask[i] == 1){
            idx++;
        }
    }
    no_survivors = idx;
    
    int *istance_numbers_seq; // tablica numeru instancji mapowanej ze starej do nowej tablicy szie: no_survivors
    istance_numbers_seq = (int*)malloc(no_survivors*sizeof(int));
    idx = 0;
    for(int i=0; i<no_instances; i++){ // to będzie sekwencyjne bo tylko przepisanie
        if(h_mask[i] == 1){
            istance_numbers_seq[idx] = i;
            idx++;
        }
    }

    no_offsprings = POPULATION_COUNT-no_survivors - MUTATION_T1 - MUTATION_T2;
    no_mutations = MUTATION_T1 + MUTATION_T2;

    hipMalloc(&d_first_pair, no_offsprings * sizeof(int));
    hipMalloc(&d_second_pair, no_offsprings * sizeof(int));
    NGrid = ceil((float)(no_offsprings)/BLOCK_SIZE);
    hipMalloc(&d_state, NGrid * BLOCK_SIZE * sizeof(hiprandState));
    initialize_rng<<<NGrid, BLOCK_SIZE>>>(d_state, time(0));
    selection_crossover_step<<<NGrid, BLOCK_SIZE>>>(d_state,d_rewards, d_first_pair, d_second_pair, no_offsprings, no_instances);
    hipFree(d_state);

    int *d_istance_numbers_seq;
    hipMalloc(&d_istance_numbers_seq, no_survivors * sizeof(int));
    hipMemcpy(d_istance_numbers_seq, istance_numbers_seq, no_survivors * sizeof(int), hipMemcpyHostToDevice);

    int *d_new_blocks_nodes;
    int *d_new_blocks_edges;
    hipMalloc(&d_new_blocks_nodes, (1+no_survivors+no_offsprings+no_mutations) * sizeof(int));
    hipMalloc(&d_new_blocks_edges, (1+no_survivors+no_offsprings+no_mutations) * sizeof(int));

    hipMemset(d_new_blocks_nodes, 0, (1+no_survivors+no_offsprings+no_mutations) * sizeof(int));
    hipMemset(d_new_blocks_edges, 0, (1+no_survivors+no_offsprings+no_mutations) * sizeof(int));

    dim3 dimGridSurvivors(ceil((float)(no_survivors)/BLOCK_SIZE),1,1);
    dim3 dimBlockSurvivors(BLOCK_SIZE,1,1);

    dim3 dimGridOffsprings(ceil((float)(no_offsprings)/BLOCK_SIZE),1,1);
    dim3 dimBlockOffsprings(BLOCK_SIZE,1,1);

    countSurvivors<<<dimGridSurvivors, dimGridOffsprings>>>(d_blocks_edges, d_blocks_nodes, d_new_blocks_edges, d_new_blocks_nodes, no_survivors, d_istance_numbers_seq); // uzupełnienie rozmiarów instancji survivors  (0 | 1 do no_survivors)

    countOffsprings<<<dimBlockOffsprings, dimBlockOffsprings>>>(d_first_pair, d_second_pair, no_offsprings, d_innov, d_blocks_edges, no_instances, d_new_blocks_edges, no_survivors + 1);
    
    countOffspringsNodes<<<dimBlockOffsprings, dimBlockOffsprings>>>(d_first_pair, d_second_pair, no_offsprings, d_translation, d_blocks_nodes, no_instances, d_new_blocks_nodes, no_survivors + 1);
    // TODO: mutations
    int *d_mutation_parent;
    d_mutation_parent = (int*)malloc((no_mutations)*sizeof(int));
    hipMalloc(&d_mutation_parent, no_mutations * sizeof(int));
    NGrid = ceil((float)(no_mutations)/BLOCK_SIZE);
    hipMalloc(&d_state, NGrid * BLOCK_SIZE * sizeof(hiprandState));
    initialize_rng<<<NGrid, BLOCK_SIZE>>>(d_state, time(0));
    countMutations<<<NGrid, BLOCK_SIZE>>>(d_state, d_mutation_parent, d_rewards , d_blocks_edges, d_blocks_nodes, d_new_blocks_edges, d_new_blocks_nodes, no_instances, no_offsprings+no_survivors);


    // end mutations
    dim3 dimGrid(ceil((float)((1+no_survivors+no_offsprings+no_mutations))/BLOCK_SIZE),1,1);
    dim3 dimBlock(BLOCK_SIZE,1,1);
    cumulatedHistogram(d_new_blocks_nodes, d_new_blocks_nodes, (1+no_survivors+no_offsprings+no_mutations));
    cumulatedHistogram(d_new_blocks_edges, d_new_blocks_edges, (1+no_survivors+no_offsprings+no_mutations));

    int *d_new_in;
    int *d_new_out;
    float *d_new_w;
    bool *d_new_enabled;
    int *d_new_innov;
    int new_no_instances; // survivors + offsprings + mutated
    int *d_new_translation; // 
    new_no_instances = no_survivors + no_offsprings + no_mutations;

    int no_edges;
    int no_nodes;

    hipMemcpy(&no_edges, d_new_blocks_edges + new_no_instances, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&no_nodes, d_new_blocks_nodes + new_no_instances, sizeof(int), hipMemcpyDeviceToHost);

    hipMalloc(&d_new_in, no_edges * sizeof(int));
    hipMalloc(&d_new_out, no_edges * sizeof(int));
    hipMalloc(&d_new_w, no_edges * sizeof(float));
    hipMalloc(&d_new_enabled, no_edges * sizeof(bool));
    hipMalloc(&d_new_innov, no_edges * sizeof(int));
    hipMalloc(&d_new_innov, no_edges * sizeof(int));
    hipMalloc(&d_new_translation, no_nodes * sizeof(int));
    
    // tytaj będzie przepisywanie survivorsów
    copySurvivors<<<dimGridSurvivors, dimGridOffsprings>>>(d_istance_numbers_seq, d_blocks_edges, d_blocks_nodes, d_new_blocks_nodes, d_new_blocks_edges, no_survivors,
    d_new_in, d_new_out, d_new_w, d_new_enabled, d_new_innov, d_new_translation, d_in, d_out, d_w, d_enabled, d_innov, d_translation);
    
    // tytaj będzie tworzenie potomków
    int *d_translation_t1;
    
    hipMalloc(&d_translation_t1, no_nodes * sizeof(int));

    int *d_translation_t2;
    

    hipMalloc(&d_translation_t2, no_nodes * sizeof(int));
    // hipMemset(d_translation_t, 0, (no_nodes) * sizeof(int));
    // printf("%d\n", no_nodes);
    crossover<<<dimBlockOffsprings, dimBlockOffsprings>>>(
    d_blocks_nodes,
    d_blocks_edges,
    d_new_blocks_nodes,
    d_new_blocks_edges,
    no_survivors, // offset
    d_in,
    d_out,
    d_w,
    d_enabled,
    d_innov,
    d_first_pair,
    d_second_pair,
    no_offsprings,
    d_new_in,
    d_new_out,
    d_new_w,
    d_new_enabled,
    d_new_innov,
    d_new_translation,
    d_translation,
    d_translation_t1,
    d_translation_t2
    );

    MutateT1T2<<<NGrid, BLOCK_SIZE>>>(
    d_state,
    d_mutation_parent,
    d_blocks_nodes,
    d_blocks_edges,
    d_new_blocks_nodes,
    d_new_blocks_edges,
    no_offsprings+no_survivors, // bardzo ważny offset
    d_in,
    d_out,
    d_w,
    d_enabled,
    d_innov,
    d_new_in,
    d_new_out,
    d_new_w,
    d_new_enabled,
    d_new_innov,
    d_new_translation,
    d_translation,
    next_edge_innov,
    next_node_innov
    );

    next_node_innov += MUTATION_T1;
    next_edge_innov += MUTATION_T1*2 + MUTATION_T2;

    hipFree(d_state);
    NGrid = ceil((float)(no_edges)/BLOCK_SIZE);
    hipMalloc(&d_state, NGrid * BLOCK_SIZE * sizeof(hiprandState));
    initialize_rng<<<NGrid, BLOCK_SIZE>>>(d_state, time(0));
    mutate_weights<<<NGrid, BLOCK_SIZE>>>(d_state, d_new_w, no_edges);
    // podmianka
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_w);
    hipFree(d_enabled);
    hipFree(d_innov);
    hipFree(d_blocks_edges);
    hipFree(d_translation);
    hipFree(d_blocks_nodes);
    hipFree(d_mask);
    hipFree(d_first_pair);
    hipFree(d_second_pair);
    *d_in_=d_new_in;
    *d_out_=d_new_out;
    *d_w_=d_new_w;
    *d_enabled_=d_new_enabled;
    *d_innov_=d_new_innov;

    *d_blocks_edges_=d_new_blocks_edges;
    *d_translation_=d_new_translation;
    *d_blocks_nodes_=d_new_blocks_nodes;
    *no_edges_ = no_edges;
    *no_nodes_ = no_nodes;
    // czyszczenie
    hipFree(d_state);
    hipFree(d_translation_t1);
    hipFree(d_translation_t2);
    hipFree(d_istance_numbers_seq);
    free(istance_numbers_seq);
    free(h_mask);

    // end czyszczenie
}

void main_loop(){
    // initial population file
    FILE *plik = fopen("crosoverCOO_test.txt", "r");
    if (plik == NULL) {
        return;
    }
    /*
    ########## wektory Populacji wejściowej ##########
    int no_instances - ilość instancji wejściowych
    int *blocks_nodes - hstogram skumulowany ilości wierzchołków (node) instancji (zaczynający się od 0) [0, end_1 + 1, end_1 + end_2 + 1, ...] (długość no_instances+1)
    int *translation - mapowanie odcinków liczb naturalnych [0,n] do rzeczywistych numerów wierzchołków   (zał że w instancjach posortowane rosnąco) (długość blocks_nodes[no_instances])
    int *blocks_edges - hstogram skumulowany ilości krawędzi (edges) instancji (długość no_instances+1)
    int *in - wejścia synaps (krawędzi/edges) instancji zmapowane do odcinka [0,n] (długość blocks_edge[no_instances])
    int *out - wejścia synaps (krawędzi/edges) instancji zmapowane do odcinka [0,n] (długość blocks_edge[no_instances])
    float *w - wagi synaps (krawędzi/edges) instancji
    bool *enabled - czy dane krawędzie są enabled (true jeżeli biorą udział w ewaluacji, false jeżeli nie biorą udziału w ewaluacji)
    int *innov - innovation number unikatowy numer rozróżniający krawędzie pomiędzy genami
    
    
    ########## wektory wejściowe z algorytmu genetycznego ##########
    int *mask - bitowa maska 0 znaczy że instancja nie przechodzi do następnej populacji 1 że przechodzi // mask which survives
    int no_survivors - ilość instancji która przetrwa (ilość jedynek w int *mask)
    int no_mutations - ilość instancji z mutacji
    int no_offsprings - ilość instancji z krzyżowania
    int *first_pair - numer pierwszego rodzica każdy index odpowiada jednemu potomkowi(długość no_offsprings)
    int *second_pair - numer drugiego rodzica każdy index odpowiada jednemu potomkowi(długość no_offsprings)
    
    ########## wektory Populacji wyjściowej (po definicje patrz "wektory Populacji wejściowej") ########## (To mamy zwrócić)
    int *new_blocks_nodes - długość no_survivors + no_offsprings + no_mutations
    int *new_blocks_edges - długość no_survivors + no_offsprings + no_mutations
    int *new_in
    int *new_out 
    float *new_w 
    bool *new_enabled 
    int *new_innov 
    int new_no_instances - długość survivors + offsprings + mutated
    int *new_translation 

    ######### Wektory pomocnicze ##########
    int *istance_numbers_seq - funkcja LUT do mapowania [0,długość survivors) w numery instancji które przechodzą do następnej populacji
    int *translation_t - tymczasowa funkcja mapująca stare indexy w nowe używana w krzyżowaniu (długość translation)
    */

    // init population:
    int *in;
    int *out;
    float *w;
    bool *enabled;
    int *innov; 

    int no_instances;
    int *blocks_edges;
    int *translation; // zał że w instancjach posortowane rosnąco
    int *blocks_nodes;
    
    // init
    fscanf(plik, "%d", &no_instances);
    blocks_nodes = (int*) malloc((no_instances+1) * sizeof(int));
    for(int i = 0; i<no_instances+1; i++){
        fscanf(plik, "%d", blocks_nodes+i);
    }
    blocks_edges = (int*) malloc((no_instances+1) * sizeof(int));
    for(int i = 0; i<no_instances+1; i++){
        fscanf(plik, "%d", blocks_edges+i);
    }
    translation = (int*) malloc((blocks_nodes[no_instances]) * sizeof(int));
    for(int i = 0; i<blocks_nodes[no_instances]; i++){
        fscanf(plik, "%d", translation+i);
    }
    innov = (int*) malloc((blocks_edges[no_instances]) * sizeof(int));
    for(int i = 0; i<blocks_edges[no_instances]; i++){
        fscanf(plik, "%d", innov+i);
    }
    enabled = (bool*) malloc((blocks_edges[no_instances]) * sizeof(int));
    for(int i = 0; i<blocks_edges[no_instances]; i++){
        int temp;
        fscanf(plik, "%d", &temp);
        *(enabled+i) = (bool)temp;
    }

    in = (int*) malloc((blocks_edges[no_instances]) * sizeof(int));
    for(int i = 0; i<blocks_edges[no_instances]; i++){
        fscanf(plik, "%d", in+i);
    }
    out = (int*) malloc((blocks_edges[no_instances]) * sizeof(int));
    for(int i = 0; i<blocks_edges[no_instances]; i++){
        fscanf(plik, "%d", out+i);
    }
    w = (float*) malloc((blocks_edges[no_instances]) * sizeof(float));
    for(int i = 0; i<blocks_edges[no_instances]; i++){
        fscanf(plik, "%f", w+i);
    }
    fclose(plik);

    // ###### inicializacja i alokacja na Device ######
    int *d_in;
    int *d_out;
    float *d_w;
    bool *d_enabled;
    int *d_innov;
    
    int *d_blocks_edges;
    int *d_translation;
    int *d_blocks_nodes;

    int *d_rewards_init;
    // alocation
    hipMalloc(&d_in, (blocks_edges[no_instances]) * sizeof(int));
    hipMalloc(&d_out, (blocks_edges[no_instances]) * sizeof(int));
    hipMalloc(&d_w, (blocks_edges[no_instances]) * sizeof(float));
    hipMalloc(&d_enabled, (blocks_edges[no_instances]) * sizeof(bool));
    hipMalloc(&d_innov, (blocks_edges[no_instances]) * sizeof(int));

    hipMalloc(&d_blocks_edges, (no_instances+1) * sizeof(int));
    hipMalloc(&d_translation, (blocks_nodes[no_instances]) * sizeof(int));
    hipMalloc(&d_blocks_nodes, (no_instances+1) * sizeof(int));

    hipMalloc(&d_rewards_init, (no_instances) * sizeof(int));

    // data copy
    hipMemcpy(d_in, in, (blocks_edges[no_instances]) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out, out, (blocks_edges[no_instances]) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_w, w, (blocks_edges[no_instances]) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_enabled, enabled, (blocks_edges[no_instances]) * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_innov, innov, (blocks_edges[no_instances]) * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_blocks_edges, blocks_edges, (no_instances+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_translation, translation, (blocks_nodes[no_instances]) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_blocks_nodes, blocks_nodes, (no_instances+1) * sizeof(int), hipMemcpyHostToDevice);

    hipMemset(d_rewards_init, 0, (no_instances) * sizeof(int));
    
    //###### inicializacja i alokacja END ######
    // ##### powielenie populacji #####
    int no_edges;
    int no_nodes;
    get_new_population(
    &d_in,
    &d_out,
    &d_w,
    &d_enabled,
    &d_innov,
    &d_blocks_edges,
    &d_translation,
    &d_blocks_nodes,
    d_rewards_init,
    no_instances,
    &no_nodes,
    &no_edges
    );


    // inicjalizacja zmiennych:
    // int *new_in;
    // int *new_out;
    // float *new_w;
    // bool *new_enabled;
    // int *new_innov;

    // int *new_translation; // 
    // int *new_blocks_nodes;
    // int new_no_instances = POPULATION_COUNT;

    // new_blocks_nodes = (int*)malloc((1+POPULATION_COUNT)*sizeof(int)); // 0 | 1 do no_survivors | no_survivors + 1 do no_survivors + no_offsprings | no_survivors + no_offsprings + 1 do no_survivors + no_offsprings + no_mutation 
     
    int *d_rewards;
    hipMalloc(&d_rewards, (POPULATION_COUNT) * sizeof(int));

    for(int i=0; i<100; i++){ // here number of iterations
        hipMemset(d_rewards, 0, (POPULATION_COUNT) * sizeof(int));
        symulate(
        d_in,
        d_out,
        d_w,
        d_enabled,
        d_innov,
        d_blocks_edges,
        d_translation,
        d_blocks_nodes,
        d_rewards,
        POPULATION_COUNT,
        no_nodes,
        no_edges,
        MAX_ITERATION_GOAL
        );

        int best_reward;
        best_reward = MAX_cryterion(d_rewards,POPULATION_COUNT);
        printf("maximum reward: %d\n", best_reward);
        if(best_reward == MAX_ITERATION_GOAL-1) break;
        get_new_population(
        &d_in,
        &d_out,
        &d_w,
        &d_enabled,
        &d_innov,
        &d_blocks_edges,
        &d_translation,
        &d_blocks_nodes,
        d_rewards,
        POPULATION_COUNT,
        &no_nodes,
        &no_edges
        );

        // new_in = (int*)malloc(no_edges*sizeof(int));
        // new_out = (int*)malloc(no_edges*sizeof(int));
        // new_w = (float*)malloc(no_edges*sizeof(float));
        // new_enabled = (bool*)malloc(no_edges*sizeof(bool));
        // new_innov = (int*)malloc(no_edges*sizeof(int));
        // new_translation = (int*)malloc(no_nodes*sizeof(int));

        // hipMemcpy(new_in, d_in, no_edges * sizeof(int), hipMemcpyDeviceToHost);
        // hipMemcpy(new_out, d_out, no_edges * sizeof(int), hipMemcpyDeviceToHost);
        // hipMemcpy(new_w, d_w, no_edges * sizeof(float), hipMemcpyDeviceToHost);
        // hipMemcpy(new_enabled, d_enabled, no_edges * sizeof(bool), hipMemcpyDeviceToHost);
        // hipMemcpy(new_innov, d_innov, no_edges * sizeof(int), hipMemcpyDeviceToHost);

        
        // hipMemcpy(new_translation, d_translation, no_nodes * sizeof(int), hipMemcpyDeviceToHost);
        // hipMemcpy(new_blocks_nodes, d_blocks_nodes, (new_no_instances+1) * sizeof(int), hipMemcpyDeviceToHost);
        // printf("\nnew block nodes: ");

        // for(int i=0; i<new_no_instances+1; i++){
        //     printf("%d\t", new_blocks_nodes[i]);
        // }
        // int *new_blocks_edges;
        // new_blocks_edges = (int*)malloc((1+POPULATION_COUNT)*sizeof(int));
        // hipMemcpy(new_blocks_edges, d_blocks_edges, (new_no_instances+1) * sizeof(int), hipMemcpyDeviceToHost);
        // printf("\nnew block edges: ");
        // for(int i=0; i<new_no_instances+1; i++){
        //     printf("%d\t", new_blocks_edges[i]);
        // }
        
        // printf("\nnew translation: ");
        // for(int i=0; i<no_nodes; i++){
        //     printf("%d\t", new_translation[i]);
        // }
        // printf("\nnew in: ");
        // for(int i=0; i<no_edges; i++){
        //     printf("%d\t", new_in[i]);
        // }

        // printf("\nnew out: ");
        // for(int i=0; i<no_edges; i++){
        //     printf("%d\t", new_out[i]);
        // }

        // printf("\nnew innov: ");
        // for(int i=0; i<no_edges; i++){
        //     printf("%d\t", new_innov[i]);
        // }
        // // printf("%d\n",i);
        // free(new_in);
        // free(new_out);
        // free(new_w);
        // free(new_enabled);
        // free(new_innov);
        // free(new_translation);
        printf("number of nodes: %d\tnumber of edges: %d\tmaximum reward: %d\n", no_nodes, no_edges, best_reward);
    }
    int copy_idx = MAX_cryterion_IDX(d_rewards, POPULATION_COUNT);
    printf("idx: %d\n", copy_idx);
    int first_node; 
    int last_node;
    int first_edge; 
    int last_edge;

    hipMemcpy(&first_edge, d_blocks_edges+copy_idx, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&last_edge, d_blocks_edges+copy_idx+1, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&first_node, d_blocks_nodes+copy_idx, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&last_node, d_blocks_nodes+copy_idx+1, sizeof(int), hipMemcpyDeviceToHost);

    int edge_size = last_edge - first_edge;
    int node_size = last_node - first_node;

    int *save_in;
    int *save_out;
    float *save_w;
    bool *save_enabled;

    save_in = (int*)malloc(edge_size*sizeof(int));
    save_out = (int*)malloc(edge_size*sizeof(int));
    save_w = (float*)malloc(edge_size*sizeof(float));
    save_enabled = (bool*)malloc(edge_size*sizeof(bool));


    hipMemcpy(save_in, d_in+first_edge, edge_size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(save_out, d_out+first_edge, edge_size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(save_w, d_w+first_edge, edge_size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(save_enabled, d_enabled+first_edge, edge_size * sizeof(bool), hipMemcpyDeviceToHost);

    plik = fopen("best_net.txt", "w");
    if (plik != NULL) {
        fprintf(plik,"%d %d\n", node_size, edge_size);
        for (int i = 0; i < edge_size; i++) {
            fprintf(plik, "%d ", save_in[i]);
        }
        fprintf(plik,"\n");
        for (int i = 0; i < edge_size; i++) {
            fprintf(plik, "%d ", save_out[i]);
        }
        fprintf(plik,"\n");
        for (int i = 0; i < edge_size; i++) {
            fprintf(plik, "%f ", save_w[i]);
        }
        fprintf(plik,"\n");
        for (int i = 0; i < edge_size; i++) {
            fprintf(plik, "%d ", (int)save_enabled[i]);
        }

        fclose(plik);
    }
    
    


    free(save_in);
    free(save_out);
    free(save_w);
    free(save_enabled);

    // printf("\nnew block nodes: ");

    // for(int i=0; i<new_no_instances+1; i++){
    //     printf("%d\t", new_blocks_nodes[i]);
    // }
    // int *new_blocks_edges;
    // new_blocks_edges = (int*)malloc((1+POPULATION_COUNT)*sizeof(int));
    // hipMemcpy(new_blocks_edges, d_blocks_edges, (new_no_instances+1) * sizeof(int), hipMemcpyDeviceToHost);
    // printf("\nnew block edges: ");
    // for(int i=0; i<new_no_instances+1; i++){
    //     printf("%d\t", new_blocks_edges[i]);
    // }
    
    // printf("\nnew translation: ");
    // for(int i=0; i<no_nodes; i++){
    //     printf("%d\t", new_translation[i]);
    // }
    // printf("\nnew in: ");
    // for(int i=0; i<no_edges; i++){
    //     printf("%d\t", new_in[i]);
    // }

    // printf("\nnew out: ");
    // for(int i=0; i<no_edges; i++){
    //     printf("%d\t", new_out[i]);
    // }

    // printf("\nnew innov: ");
    // for(int i=0; i<no_edges; i++){
    //     printf("%d\t", new_innov[i]);
    // }

    // printf("\nnew enabled: ");
    // for(int i=0; i<no_edges; i++){
    //     printf("%d\t", (int)new_enabled[i]);
    // }
    // printf("\n");

    // tutaj free TODO:
    hipFree(d_rewards);
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_w);
    hipFree(d_enabled);
    hipFree(d_innov);
    hipFree(d_blocks_edges);
    hipFree(d_blocks_nodes);
    hipFree(d_translation);
    hipFree(d_rewards_init);

    free(blocks_nodes);
    free(blocks_edges);
    free(translation);
    free(innov);
    free(enabled);
    free(in);
    free(out);
    free(w);
}




int main(){
    main_loop();
}